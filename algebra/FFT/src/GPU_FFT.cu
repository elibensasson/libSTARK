#include "hip/hip_runtime.h"
/*

 * GPU_FFT.cu
 *
 *  Created on: Aug 26, 2014
 *      Author: matan
 */
#include "GPU_FFT.cuh"
#include "Polynomials.h"
#include "Definitions.h"
#include <ctime>
#ifdef __GNUC__
#include <sys/time.h>
#endif	//#ifdef __GNUC__
#include <iostream>
#include <stdio.h>
//#include "Tests.h"
#include "Chunk.cuh"
#include <bitset>
#include <iostream>
#include <cstdio>

using namespace std;
#ifdef __GPU
namespace FFF{
/*------- CHUNK ------*/
/*
 * Chunk constant memory
 */
__constant__ idx_t p_Mod[max_nonzero_coefs_in_mod];
__constant__ idx_t p_ModLen;
__constant__ Element element_mul;
__constant__ Chunk c;

/*
 * FFT Constant Memory
 */
__constant__ chunk_cell_t d_chunk_cell_mask[Chunk::log_elements_in_chunk+2];
__constant__ chunk_cell_t d_partition[1<<bits_in_byte];
__constant__ chunk_cell_t d_alter_mask[Chunk::log_elements_in_chunk+1];
__constant__ Chunk d_linear_mul[1];
__constant__ Chunk d_ilinear_mul[1];

#define DUP_ODD_BITS(cell) cell = ((cell) & 0x55555555) | (((cell)&0x55555555)<<1)
#define DUP_EVEN_BITS(cell) cell = ((cell) & 0xaaaaaaaa) | (((cell)&0xaaaaaaaa)>>1)
__device__ void a_chunkToNormal(Chunk *d_a, Elements_Chunk *d_b, idx_t idx)
{
	cell_t ans = 0;
	idx_t element_idx = idx & andMask(Chunk::log_elements_in_chunk);
	idx_t cell_idx = idx >> Chunk::log_elements_in_chunk;
	for(unsigned int i = cell_idx<<Element::log_bits_in_cell ; i < ((cell_idx+1)<<Element::log_bits_in_cell); ++i)
		ans^=(((cell_t)(((d_a->v[i])>>(element_idx))&1))<<(i-(cell_idx<<Element::log_bits_in_cell)));
	d_b->e[element_idx].c[cell_idx]=ans;
}
__global__ void k_chunkToNormal(Chunk *d_a,Elements_Chunk *d_b , len_t len)
{
	const unsigned int threads_in_chunk = Chunk::elements_in_chunk * Element::element_len;
	 __shared__ Chunk input[max_block_size / threads_in_chunk];
	idx_t idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= len*threads_in_chunk)
		return;
	idx_t chunkIdx = (idx) / (Element::element_len*Chunk::elements_in_chunk);
	idx_t in_chunkIdx = (idx  & (Element::element_len * Chunk::elements_in_chunk - 1));
	idx_t chunks_in_block = blockDim.x / Chunk::cells_in_chunk;
	idx_t inBlockChunkIdx = chunkIdx & (threads_in_chunk-1);
	for(unsigned int i = 0 ; i < sizeof(cell_t)/sizeof(chunk_cell_t) ; ++i){
		input[inBlockChunkIdx].v[in_chunkIdx + i*threads_in_chunk] = d_a[chunkIdx].v[in_chunkIdx+i*threads_in_chunk];
	}
	a_chunkToNormal(&(input[inBlockChunkIdx]), &(d_b[chunkIdx]),in_chunkIdx);
}
__host__	void Chunk::chunkToNormal(Chunk(*h_a), Elements_Chunk(*h_b), len_t len, bool copy)
{
	//Declare device variables
	Chunk (*d_a);
	Elements_Chunk (*d_b);

	const unsigned int num_element = len*elements_in_chunk;
	const unsigned int threads = Element::element_len * num_element;

	//Define Block  and Grid Size.
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
	if(copy){
	//Allocate Memory on GPU. (global)
		hipMalloc(&d_a,sizeof(Chunk)*len);
		hipMalloc(&d_b,sizeof(Elements_Chunk)*len);

	//Copy memory to GPU.
		hipMemcpy(d_a,h_a,sizeof(Chunk)*len,hipMemcpyHostToDevice);
	} else {
		d_a = h_a;
		d_b = h_b;
	}

	//Launch Kernel
	k_chunkToNormal<<<gridSize,blockSize>>>(d_a,d_b,len);
	if(copy){
	//Copy results back to memory
		hipMemcpy(h_b,d_b,sizeof(Elements_Chunk)*len,hipMemcpyDeviceToHost);

	//Free allocated memory.
		hipFree(d_a);
		hipFree(d_b);
	}
}
__device__ void a_normalToChunk(Elements_Chunk *d_a, Chunk *d_b, idx_t idx)
{
	chunk_cell_t ans = 0;
	idx_t cell_idx = idx>>Element::log_bits_in_cell;
	for(unsigned int i = 0 ; i < Chunk::elements_in_chunk ; ++i)
		ans^=((((d_a->e[i].c[cell_idx])>>(idx& andMask(Element::log_bits_in_cell)))&1)<<i);
	d_b->v[idx]=ans;
}
__global__		void k_normalToChunk(Elements_Chunk *d_a,Chunk *d_b , len_t len)
{
	idx_t idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= (len<<Chunk::log_cells_in_chunk))
		return;
	idx_t chunkIdx = (idx) >> Chunk::log_cells_in_chunk;
	idx_t in_chunkIdx = (idx  & andMask(Chunk::log_cells_in_chunk));
	a_normalToChunk(&(d_a[chunkIdx]),&(d_b[chunkIdx]),in_chunkIdx);
}
__host__	void Chunk::normalToChunk(Elements_Chunk(*h_a), Chunk (*h_b), len_t len,bool copy)
{

	//Declare device variables
	Elements_Chunk (*d_a);
	Chunk (*d_b);

	const unsigned int threads = len<<Chunk::log_cells_in_chunk;

	//Define Block  and Grid Size.
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);

	//Allocate Memory on GPU. (global)
	if(copy){
		hipMalloc(&d_a,sizeof(Elements_Chunk)*len);
		hipMalloc(&d_b,sizeof(Chunk)*len);

	//Copy memory to GPU.
		hipMemcpy(d_a,h_a,sizeof(Elements_Chunk)*len,hipMemcpyHostToDevice);
	} else{
		d_a = h_a;
		d_b = h_b;
	}
	//Launch Kernel
	k_normalToChunk<<<gridSize,blockSize>>>(d_a,d_b,len);

	//Copy results back to memory
	if(copy){
		hipMemcpy(h_b,d_b,sizeof(Chunk)*len,hipMemcpyDeviceToHost);
		//Free allocated memory.
		hipFree(d_a);
		hipFree(d_b);
	}

}
__host__ 	void Chunk::setMod(){
	hipMemcpyToSymbol(HIP_SYMBOL(p_Mod),&(Element::irr_poly_index[ord>>log_warp_size]),sizeof(idx_t)*max_nonzero_coefs_in_mod);
	hipMemcpyToSymbol(HIP_SYMBOL(p_ModLen),&(Element::mod_len[ord>>log_warp_size]),sizeof(idx_t));
}
//__device__ void Chunk::chunk_reduce_xor(Chunk *a, Chunk *c_bottom, Chunk*c_top, idx_t idx)
//{
//	chunk_cell_t ans=c_bottom->v[idx];
//	unsigned int temp_idx;
//	for(idx_t i = 0 ; i < p_ModLen ; ++i)
//	{
//		for(idx_t j = 0 ; j < p_ModLen ; ++j)
//		{
//			temp_idx = idx+(ord<<1)-p_Mod[i]-p_Mod[j];
//			if(temp_idx >= (ord<<1)-p_Mod[j] && temp_idx < (ord<<1))
//				ans^=c_top->v[temp_idx-ord];
//		}
//	}
//	a->v[idx]^=ans;
//}
__device__ void Chunk::chunk_xor(Chunk *a, Chunk* b, idx_t idx){
	a->v[idx]^=b->v[idx];
}
__device__ void Chunk::chunk_reduce_xor(Chunk *a, Chunk *c_bottom, idx_t idx,Chunk* to_xor ,int shift)
{
	// replaced p_ModLen-1 by 4
	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
#pragma unroll
		for(unsigned int j = 0 ; j < 4 ; ++j)
		{
			c_bottom->v[(ord>>1)+idx+i+p_Mod[j]]^=c_bottom->v[(ord>>1)+ord+idx+i];
		}
	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
#pragma unroll
		for(unsigned int j = 0 ; (j) < 4 ; ++j)
		{
			c_bottom->v[idx+i+p_Mod[j]]^=c_bottom->v[ord+idx+i];
		}
		for(unsigned int i = 0 ; i < ord ; i+=warp_size){
			to_xor->v[idx+i]^=(c_bottom->v[idx+i]>>shift);
		}
}

__device__ void Chunk::chunk_reduce(Chunk *a, Chunk *c_bottom, idx_t idx)
{
	//replaced p_ModLen with 5
	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
		for(unsigned int j = 0 ; j+1 < 5 ; ++j)
		{
			c_bottom->v[(ord>>1)+idx+i+p_Mod[j]]^=c_bottom->v[(ord>>1)+ord+idx+i];
		}
	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
		for(unsigned int j = 0 ; (j+1) < 5 ; ++j)
		{
			c_bottom->v[idx+i+p_Mod[j]]^=c_bottom->v[ord+idx+i];
		}
		for(unsigned int i = 0 ; i < ord ; i+=warp_size){
			a->v[idx+i]=c_bottom->v[idx+i];
		}
}
__device__ void Chunk::chunkClmul(Chunk (*a),  Element (*e), idx_t idx, Chunk (*c))
{
	chunk_cell_t my_ans[2][(ord>>(log_warp_size))]={0};
	for(unsigned int k = 0 ; k < ord ; ++k)
	{
		if(EXTRACT_BIT(e->c,k))
			for(unsigned int t = 0 ; t <  (ord>>log_warp_size); ++t)
			{
				int b = (k>(idx+warp_size*t));
				my_ans[b][t]^=a->v[idx+warp_size*t+(b<<log_ord)-k];
			}
	}
	for(unsigned int i = 0 ; i < (ord>>log_warp_size); ++i)
	{
		c->v[idx+i*warp_size] = my_ans[0][i];
		c->v[ord+idx+i*warp_size] = my_ans[1][i];
	}
}
__device__ void Chunk::aux_k_clmul(Chunk *a, Element* e, len_t len,Chunk* c_shared)
{

	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size)
		my_shared_chunk->v[in_chunk_idx+i]=a[chunk_idx].v[in_chunk_idx+i];
	Chunk::chunkClmul(my_shared_chunk,e,in_chunk_idx,my_shared_chunk);
	Chunk::chunk_reduce(a+chunk_idx,my_shared_chunk,in_chunk_idx);
}
__global__ void k_clmul(Chunk *a,Element *e,len_t len )
{
	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
	__shared__ Chunk c_shared[shared_len<<1];
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=(len<<Chunk::log_threads_in_chunk))
		return;
	Chunk::aux_k_clmul(a,e,len,c_shared);
}
__host__	void Chunk::mul(Chunk (*h_a),Element (*h_e),len_t len, Chunk (*h_res)){

#ifdef __MEASURE
	hipEvent_t start,stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif
	//Declare device variables
	Chunk 	(*d_a);
	Element (*d_e);

	//Define Block  and Grid Size.
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(len<<Chunk::log_threads_in_chunk,max_block_size),1,1);

	//Allocate Memory on GPU. (global)
	hipMalloc(&d_a,sizeof(Chunk)*len);
	hipMalloc(&d_e,sizeof(Element));

	//Copy memory to GPU.
	hipMemcpy(d_a,h_a,sizeof(Chunk)*len,hipMemcpyHostToDevice);
	hipMemcpy(d_e,h_e,sizeof(Element),hipMemcpyHostToDevice);

	//Set Mod
	setMod();
//	setElementMul(h_e);
	//Launch Kernel
#ifdef __MEASURE
	hipEventRecord(start,0);
#endif
	k_clmul<<<gridSize,blockSize>>>(d_a,d_e,len);
#ifdef __MEASURE
	hipEventRecord(stop,0);
#endif

	//Copy results to host
	hipMemcpy(h_res,d_a,sizeof(Chunk)*len,hipMemcpyDeviceToHost);
	//Free allocated memory.
	hipFree(d_a);
	hipFree(d_e);
#ifdef __MEASURE
	hipEventElapsedTime(&time,start,stop);
	printf("Time for the mul: %f ms on %d chunks \n",time,len);
#endif
}

__global__ void k_add(Chunk (*a), Chunk (*b), len_t l)
{
	unsigned int idx = threadIdx.x+blockIdx.x*blockDim.x;
	if(idx>=l*Chunk::cells_in_chunk)
		return;
	((chunk_cell_t*)a)[idx]^=((chunk_cell_t*)b)[idx];
}
__host__ void Chunk::add(Chunk (*h_a),Chunk (*h_b),len_t len)
{

	//Declare device variables
	Chunk (*d_a);
	Chunk (*d_b);

	//Define Block  and Grid Size.
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(max_block_size,len),1,1);

	//Allocate Memory on GPU. (global)
	hipMalloc(&d_a,sizeof(Chunk)*len);
	hipMalloc(&d_b,sizeof(Chunk)*len);

	//Copy memory to GPU.
	hipMemcpy(d_a,h_a,sizeof(Chunk)*len,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(Chunk)*len,hipMemcpyHostToDevice);

	//Launch Kernel
	k_add<<<gridSize,blockSize>>>(d_a,d_b,len);

	//Copy results to CPU memory
	hipMemcpy(h_a,d_a,sizeof(Chunk)*len,hipMemcpyDeviceToHost);

	//Free allocated memory.
	hipFree(d_a);
	hipFree(d_b);
}
__host__ void Chunk::print() const {
	for(unsigned int i = 0 ; i < cells_in_chunk ; ++i){
		cout << bitset<bits_in_byte*sizeof(chunk_cell_t)>(this->v[i])<<endl;
	}
}
__host__	void Elements_Chunk::print() const{
		for(unsigned int i = 0 ; i < elements_in_elements_chunk ; ++i){
			Element::printElement(this->e[i]);
			cout<<endl;
		}
	}
//Mul chunk by another chunk
__device__ void Chunk::clmul_by_chunk_bShuffle_ixor_mask(const Chunk& a,const  Chunk& e, const idx_t in_chunk_idx, Chunk& c, const int shift, const idx_t mask_idx){
	/*
	 * Carryles multiplication
	 */
	chunk_cell_t a_reg[2];
	chunk_cell_t e_reg;
	a_reg[0] = a.v[in_chunk_idx];
	a_reg[1] = a.v[in_chunk_idx+warp_size];

	a_reg[0]^=(a_reg[0]&d_alter_mask[mask_idx+1])<<(1<<mask_idx);
	a_reg[1]^=(a_reg[1]&d_alter_mask[mask_idx+1])<<(1<<mask_idx);

	chunk_cell_t my_ans[2][2]={0};
	int b;
	for(unsigned k = 0 ; k < warp_size; ++k){
		e_reg=e.v[k];
		b= (in_chunk_idx>= k);
		my_ans[0][0] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[0][1] ^= ((1-b)*__shfl_down(a_reg[0],warp_size-k))& e_reg;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;

		e_reg=e.v[k+warp_size];;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[0],32-k))& e_reg;
		my_ans[1][0] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][1] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;
	}
	/*
	 * Reduce
	 */
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[1][0]^=(b*__shfl_down(my_ans[1][1],32-p_Mod[i]));
		my_ans[0][1]^=((1-b)*__shfl_up(my_ans[1][1],p_Mod[i]));
	}
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[0][1]^=(b * __shfl_down(my_ans[1][0],32-p_Mod[i]));
		my_ans[0][0]^=((1-b) * __shfl_up(my_ans[1][0],p_Mod[i]));
	}

	a_reg[0] ^= (my_ans[0][0] >> shift);
	a_reg[1] ^= (my_ans[0][1] >> shift);

	c.v[in_chunk_idx]  = a_reg[0];
	c.v[in_chunk_idx+warp_size] = a_reg[1];
}
__device__ void Chunk::clmul_by_chunk_bShuffle_xor_mask(const Chunk& a,const  Chunk& e, const idx_t in_chunk_idx, Chunk& c, const int shift, const idx_t mask_idx){
	/*
	 * Carryles multiplication
	 */
	chunk_cell_t a_reg[2];
	chunk_cell_t e_reg;
	a_reg[0] = a.v[in_chunk_idx];
	a_reg[1] = a.v[in_chunk_idx+warp_size];

	chunk_cell_t my_ans[2][2]={0};
	int b;
	for(unsigned k = 0 ; k < warp_size; ++k){
		e_reg=e.v[k];
		b= (in_chunk_idx>= k);
		my_ans[0][0] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[0][1] ^= ((1-b)*__shfl_down(a_reg[0],warp_size-k))& e_reg;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;

		e_reg=e.v[k+warp_size];;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[0],32-k))& e_reg;
		my_ans[1][0] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][1] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;
	}
	/*
	 * Reduce
	 */
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[1][0]^=(b*__shfl_down(my_ans[1][1],32-p_Mod[i]));
		my_ans[0][1]^=((1-b)*__shfl_up(my_ans[1][1],p_Mod[i]));
	}
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[0][1]^=(b * __shfl_down(my_ans[1][0],32-p_Mod[i]));
		my_ans[0][0]^=((1-b) * __shfl_up(my_ans[1][0],p_Mod[i]));
	}

	a_reg[0] ^= (my_ans[0][0] >> shift);
	a_reg[1] ^= (my_ans[0][1] >> shift);

	a_reg[0]^=(a_reg[0]&d_alter_mask[mask_idx+1])<<(1<<mask_idx);
	a_reg[1]^=(a_reg[1]&d_alter_mask[mask_idx+1])<<(1<<mask_idx);

	c.v[in_chunk_idx]  = a_reg[0];
	c.v[in_chunk_idx+warp_size] = a_reg[1];
}
__device__ void Chunk::clmul_by_chunk_bShuffle_xor(const Chunk& a,const  Chunk& e, const idx_t in_chunk_idx, Chunk& c, const int shift){
	/*
	 * Carryles multiplication
	 */
	chunk_cell_t a_reg[2];
	chunk_cell_t e_reg;
	a_reg[0] = a.v[in_chunk_idx];
	a_reg[1] = a.v[in_chunk_idx+warp_size];

	chunk_cell_t my_ans[2][2]={0};
	int b;
	for(unsigned k = 0 ; k < warp_size; ++k){
		e_reg=e.v[k];
		b= (in_chunk_idx>= k);
		my_ans[0][0] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[0][1] ^= ((1-b)*__shfl_down(a_reg[0],warp_size-k))& e_reg;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;

		e_reg=e.v[k+warp_size];;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[0],32-k))& e_reg;
		my_ans[1][0] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][1] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;
	}
	/*
	 * Reduce
	 */
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[1][0]^=(b*__shfl_down(my_ans[1][1],32-p_Mod[i]));
		my_ans[0][1]^=((1-b)*__shfl_up(my_ans[1][1],p_Mod[i]));
	}
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[0][1]^=(b * __shfl_down(my_ans[1][0],32-p_Mod[i]));
		my_ans[0][0]^=((1-b) * __shfl_up(my_ans[1][0],p_Mod[i]));
	}
		c.v[in_chunk_idx] ^= (my_ans[0][0] >> shift);
		c.v[in_chunk_idx+warp_size] ^= (my_ans[0][1] >> shift);
}
__device__ void Chunk::clmul_by_chunk_bShuffle(const Chunk& a,const  Chunk& e, const idx_t in_chunk_idx, Chunk& c){
	/*
	 * Carryles multiplication
	 */
	chunk_cell_t a_reg[2];
	chunk_cell_t e_reg;
	a_reg[0] = a.v[in_chunk_idx];
	a_reg[1] = a.v[in_chunk_idx+warp_size];

	chunk_cell_t my_ans[2][2]={0};
	int b;
	for(unsigned k = 0 ; k < warp_size; ++k){
		e_reg=e.v[k];
		b= (in_chunk_idx>= k);
		my_ans[0][0] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[0][1] ^= ((1-b)*__shfl_down(a_reg[0],warp_size-k))& e_reg;
		my_ans[0][1] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;

		e_reg=e.v[k+warp_size];
		my_ans[0][1] ^= (b*__shfl_up(a_reg[0],k)) & e_reg;
		my_ans[1][0] ^= ((1-b)*__shfl_down(a_reg[0],32-k))& e_reg;
		my_ans[1][0] ^= (b*__shfl_up(a_reg[1],k)) & e_reg;
		my_ans[1][1] ^= ((1-b)*__shfl_down(a_reg[1],32-k))& e_reg;
	}
	/*
	 * Reduce
	 */
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[1][0]^=(b*__shfl_down(my_ans[1][1],32-p_Mod[i]));
		my_ans[0][1]^=((1-b)*__shfl_up(my_ans[1][1],p_Mod[i]));
	}
#pragma unroll 4
	for(unsigned int i = 0 ; i < 4 ; ++i){
		b=(in_chunk_idx<p_Mod[i]);
		my_ans[0][1]^=(b * __shfl_down(my_ans[1][0],32-p_Mod[i]));
		my_ans[0][0]^=((1-b) * __shfl_up(my_ans[1][0],p_Mod[i]));
	}
		c.v[in_chunk_idx] = my_ans[0][0];
		c.v[in_chunk_idx+warp_size] = my_ans[0][1];
}
__device__ void Chunk::clmul_by_chunk(Chunk& a, Chunk& e, idx_t idx, Chunk* c){
	chunk_cell_t my_ans[2][(ord>>(log_warp_size))]={0};
	int b;
//	const int l = ord>>log_warp_size;
	for(unsigned int k = 0 ; k < ord ; ++k)
#pragma unroll 2
		for(unsigned int t = 0 ; t <  2; ++t)
		{
			b = (k>(idx+warp_size*t));
			my_ans[b][t]^=a.v[idx+warp_size*t+(b<<log_ord)-k]&e.v[k];
		}
#pragma unroll 2
	for(unsigned int i = 0 ; i < 2; ++i)
	{
		c->v[idx+i*warp_size] = my_ans[0][i];
		c->v[ord+idx+i*warp_size] = my_ans[1][i];
	}
}
__global__ void k_mul_chunk(Chunk* cs, Chunk* c, len_t cs_len)
{
	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
	__shared__ Chunk c_shared[shared_len<<1];
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=(cs_len<<Chunk::log_threads_in_chunk))
		return;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
		my_shared_chunk->v[in_chunk_idx+i]=cs[chunk_idx].v[in_chunk_idx+i];
		my_shared_chunk[1].v[in_chunk_idx+i]=c->v[in_chunk_idx+i];
	}
	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
	Chunk::chunk_reduce(cs+chunk_idx,my_shared_chunk,in_chunk_idx);
}
__global__ void k_mul_chunk_xor(Chunk* cs, Chunk* c, len_t cs_len,Chunk* to_xor, int shift = 0)
{
	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
	__shared__ Chunk c_shared[shared_len<<1];
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=(cs_len<<Chunk::log_threads_in_chunk))
		return;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
		my_shared_chunk->v[in_chunk_idx+i]=cs[chunk_idx].v[in_chunk_idx+i];
		my_shared_chunk[1].v[in_chunk_idx+i]=c->v[in_chunk_idx+i];
	}
	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
	Chunk::chunk_reduce_xor(cs+chunk_idx,my_shared_chunk,in_chunk_idx,to_xor,shift);
}
//Mul a chunk by a chunk
void Chunk::chunk_mul(Chunk (* h_a), Chunk (*h_b) , len_t len, Chunk (*h_res), bool copy, bool do_xor, int shift){
#ifdef __MEASURE
	hipEvent_t start,stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif
	//Declare device variables
	Chunk 	(*d_a);
	Chunk   (*d_b);

	//Define Block  and Grid Size.
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(len<<Chunk::log_threads_in_chunk,max_block_size),1,1);

	if(copy){
	//Allocate Memory on GPU. (global)
		hipMalloc(&d_a,sizeof(Chunk)*len);
		hipMalloc(&d_b,sizeof(Chunk));
	//Copy memory to GPU.
		hipMemcpy(d_a,h_a,sizeof(Chunk)*len,hipMemcpyHostToDevice);
		hipMemcpy(d_b,h_b,sizeof(Chunk),hipMemcpyHostToDevice);
	} else {
		d_a = h_a;
		d_b = h_b;
	}


	//Set Mod
	setMod();
//	setElementMul(h_e);
	//Launch Kernel
#ifdef __MEASURE
	hipEventRecord(start,0);
#endif
	if(do_xor)
		k_mul_chunk_xor<<<gridSize,blockSize>>>(d_a,d_b,len,d_a,shift);
	else
		k_mul_chunk<<<gridSize,blockSize>>>(d_a,d_b,len);
#ifdef __MEASURE
	hipEventRecord(stop,0);
#endif

	if(copy){
	//Copy results to host
		hipMemcpy(h_res,d_a,sizeof(Chunk)*len,hipMemcpyDeviceToHost);
	//Free allocated memory.
		hipFree(d_a);
		hipFree(d_b);
	}
#ifdef __MEASURE
	hipEventElapsedTime(&time,start,stop);
	printf("Time for the mul: %f ms on %d chunks \n",time,len);
#endif
}









/*-------------------------------------*/

/*** GPU FFT ***/
const unsigned int multThreadsInBlock = 1024;

__global__ void k_multiExp_mult_bShuffle(Chunk* d_a, Chunk* d_b , len_t b_len , len_t a_len){
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=(a_len<<Chunk::log_threads_in_chunk))
		return;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
//	const idx_t shared_chunk_idx = ;
	Chunk::clmul_by_chunk_bShuffle(d_a[chunk_idx],d_b[chunk_idx & (b_len-1)],in_chunk_idx,d_a[chunk_idx]);
//	Chunk::chunk_reduce(d_a+chunk_idx,my_shared_chunk,in_chunk_idx);
}
__global__ void k_multiExp_mult(Chunk* d_a, Chunk* d_b , len_t b_len , len_t a_len){
//	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
//	__shared__ Chunk c_shared[shared_len<<1];
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=(a_len<<Chunk::log_threads_in_chunk))
		return;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
//	const idx_t shared_chunk_idx = ;
//	Chunk* my_shared_chunk = c_shared+(((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk))<<1);
//	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
//		my_shared_chunk->v[in_chunk_idx+i]=d_a[chunk_idx].v[in_chunk_idx+i];
//		my_shared_chunk[1].v[in_chunk_idx+i]=d_b[chunk_idx & (b_len - 1)].v[in_chunk_idx+i];
//	}
	Chunk::clmul_by_chunk_bShuffle(d_a[chunk_idx],d_b[chunk_idx & (b_len - 1)],in_chunk_idx,d_a[chunk_idx]);
//	Chunk::chunk_reduce(d_a+chunk_idx,my_shared_chunk,in_chunk_idx);
}

/*
 * Gets as input:
 * 	d_a - the polynomial.
 * 	d_b - the multiexponent of current b_m for on strip of elements.
 * 	b_len - length of d_b in chunks.
 *
 * 	Multiplies chunk d_a[i] by chunk d_b[i mod b_len].
 */
void GPU_FFT::multiExp_mult(len_t a_len, Chunk* d_a , Chunk* d_b , len_t b_len){
	unsigned int threads = (a_len<<Chunk::log_elements_in_chunk);
	dim3 blockSize(multThreadsInBlock,1,1);
	dim3 gridSize(sizeCiel(threads,multThreadsInBlock),1,1);

	//Launch Kernel
	k_multiExp_mult_bShuffle<<<gridSize,blockSize>>>(d_a,d_b,b_len, threads>>Chunk::log_elements_in_chunk);
}

void GPU_FFT::multiExponentiate_gpu(const FFT* fft,Chunk* d_chunk_P, len_t p_len, len_t dim, Chunk* d_exp ){
	//Copy exps to memory.
	if(dim<=Chunk::log_elements_in_chunk){
		hipMemcpy(d_exp,fft->gpu_exp[fft->basis.getSize()-dim],
				sizeof(Chunk),hipMemcpyHostToDevice);
		multiExp_mult(p_len,d_chunk_P,d_exp,1);
	} else {
		hipMemcpy(d_exp,fft->gpu_exp[fft->basis.getSize()-dim],
				sizeof(Chunk)*(1<<(dim-Chunk::log_elements_in_chunk)),hipMemcpyHostToDevice);
		multiExp_mult(p_len,d_chunk_P,d_exp,1<<(dim-Chunk::log_elements_in_chunk));
	}

}

void GPU_FFT::imultiExponentiate_gpu(const FFT* fft,Chunk* d_chunk_P, len_t p_len, len_t dim, Chunk* d_exp ){
	//Copy exps to memory.
	if(dim<=Chunk::log_elements_in_chunk){
		hipMemcpy(d_exp,fft->gpu_i_exp[fft->basis.getSize()-dim],
				sizeof(Chunk),hipMemcpyHostToDevice);
		multiExp_mult(p_len,d_chunk_P,d_exp,1);
	} else {
		hipMemcpy(d_exp,fft->gpu_i_exp[fft->basis.getSize()-dim],
				sizeof(Chunk)*(1<<(dim-Chunk::log_elements_in_chunk)),hipMemcpyHostToDevice);
		multiExp_mult(p_len,d_chunk_P,d_exp,1<<(dim-Chunk::log_elements_in_chunk));
	}

}

__device__ void taylor_smaller_than_chunk(Chunk* chunk,idx_t in_chunk_idx, len_t t_dim, len_t p_len, idx_t idx){
		if(idx >= ((p_len) << (Chunk::log_cells_in_chunk)))
			return;
		//Performs the rest of the expansion.
		chunk_cell_t cell = chunk->v[in_chunk_idx];
		for(; t_dim >=2 ; --t_dim){
			cell ^= (cell & (d_chunk_cell_mask[t_dim]<<((1<<t_dim)-(1<<(t_dim-2)))))>>(1<<(t_dim-2));
			cell ^= (cell & (d_chunk_cell_mask[t_dim]<<(1<<(t_dim-1))))>>(1<<(t_dim-2));
		}
		chunk->v[in_chunk_idx]= cell;
}
__global__ void k_taylorExpansion_iteration_large(Chunk* d_chunk_P , len_t p_len , len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	idx_t in_chunk_idx = idx & (Chunk::cells_in_chunk-1);
	/*
	 * flag = true if current sub-polynomial fits in a thread block.
	 */
	bool flag = (t_dim-Chunk::log_elements_in_chunk <= log_max_block_size-Chunk::log_cells_in_chunk + 2);
	if(idx >= ((p_len) << (Chunk::log_cells_in_chunk-2)))
		return;
	do {
		len_t sub_len = (1<<(t_dim-Chunk::log_elements_in_chunk));
		idx_t chunk_idx = idx >> (Chunk::log_cells_in_chunk);
		chunk_idx = (chunk_idx /(sub_len>>2))*(sub_len) + ((chunk_idx) & ((sub_len>>2)-1));
		d_chunk_P[(sub_len>>1) + chunk_idx ].v[in_chunk_idx] ^= d_chunk_P[(3*(sub_len>>2)) + chunk_idx].v[in_chunk_idx];
		d_chunk_P[(sub_len>>2) + chunk_idx ].v[in_chunk_idx] ^= d_chunk_P[(sub_len>>1) + chunk_idx].v[in_chunk_idx];
		if(flag)
			__syncthreads();
		--t_dim;
	} while ( flag && t_dim > Chunk::log_elements_in_chunk + 1);
		//If number of threads needed is less than a thread block - we can just continue!
}
__global__ void k_taylorExpansion_iteration_twoChunks(Chunk* d_chunk_P, len_t p_len, len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	idx_t in_chunk_idx = idx & (Chunk::cells_in_chunk-1);
	if(idx >= ((p_len) << (Chunk::log_cells_in_chunk-1)))
		return;
	idx_t chunk_idx= (idx >> (Chunk::log_cells_in_chunk) ) << 1;
	d_chunk_P[(chunk_idx)+1].v[in_chunk_idx]^= (d_chunk_P[(chunk_idx)+1].v[in_chunk_idx]>>(Chunk::elements_in_chunk>>1));
	d_chunk_P[(chunk_idx)].v[in_chunk_idx]^= (d_chunk_P[(chunk_idx)+1].v[in_chunk_idx]<<(Chunk::elements_in_chunk>>1));
}
__global__ void k_taylorExpansion_iteration_singleChunk(Chunk* d_chunk_P, len_t p_len, len_t t_dim){

	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	idx_t in_chunk_idx = idx & (Chunk::cells_in_chunk-1);
	//Performs the rest of the expansion.
	idx_t chunk_idx = (idx >> (Chunk::log_cells_in_chunk));
	taylor_smaller_than_chunk(d_chunk_P+chunk_idx,in_chunk_idx,t_dim,p_len,idx);
}
void taylorExpansion_iteration(const FFT* fft, Chunk * d_chunk_P , len_t p_len , len_t t_dim){
	unsigned int threads;
	dim3 blockSize(max_block_size,1,1);
	if( t_dim >= Chunk::log_elements_in_chunk + 2){
		threads = p_len<<(Chunk::log_cells_in_chunk-2);
		dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
		k_taylorExpansion_iteration_large<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
	} else if (t_dim == Chunk::log_elements_in_chunk + 1){
		threads = p_len <<(Chunk::log_cells_in_chunk-1);
		dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
		k_taylorExpansion_iteration_twoChunks<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
	} else {
		threads = p_len << Chunk::log_cells_in_chunk;
		dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
		k_taylorExpansion_iteration_singleChunk<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
	}
//	k_taylorExpansion_iteration<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
}
void GPU_FFT::taylorExpansion_gpu(const FFT* fft, Chunk* d_chunk_P, len_t p_len , len_t dim){
	len_t t_dim = dim;
	/*
	 * The sub-polynomial requires more than a single thread block.
	 */
	while(t_dim+Chunk::log_cells_in_chunk > Chunk::log_elements_in_chunk+log_max_block_size + 2){
		taylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim);
		--t_dim;
	}
	/*
	 * The sub-polynomial requires more than 2 chunks BUT less than a single thread block.
	 */
	if(t_dim >= Chunk::log_elements_in_chunk + 2){
		taylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim);
		t_dim = Chunk::log_elements_in_chunk+1;
	}
	/*
	 * Each sub-polynomial takes exactly two chunks.
	 */
	if(t_dim == Chunk::log_elements_in_chunk + 1){
		taylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim);
		--t_dim;
	}
	/*
	 * Each sub-polynomial takes at most one chunk.
	 */
	taylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim);
}

	/*
	 *  Input:
	 *  1) d_chunk_p - The polynomial on device's memory.
	 *  2) p_len - number of chunks in d_chunk_p.
	 *  3) chunk_idx - the chunk current thread has to deal with.
	 *  4) in_chunk_idx - the number of the cell the current thread deals with.
	 *  5) t_dim - The dim of the original (input) subpolynomial.
	 *
	 *  This function performs what a single thread does when performing the partition function, on a single chunk.
	 */
__device__ void partition_in_chunk(Chunk* d_chunk_p, len_t p_len , idx_t in_chunk_idx, len_t t_dim){
	chunk_cell_t ans = d_chunk_p->v[in_chunk_idx];
	for(unsigned int i = 2 ; i <= Chunk::log_elements_in_chunk && i<=t_dim ; ++i){
		ans = 	(ans & d_chunk_cell_mask[i]) |
				(ans & (d_chunk_cell_mask[i]<<((1<<(i))-(1<<(i-2))))) |
				(ans & (d_chunk_cell_mask[i]<<(1<<(i-1))))>>(1<<(i-2)) |
				(ans & (d_chunk_cell_mask[i]<<(1<<(i-2))))<<(1<<(i-2));
	}
	d_chunk_p->v[in_chunk_idx]=ans;
}

__device__ void partition_two_chunks(Chunk* d_chunk_p, idx_t in_chunk_idx){
	chunk_cell_t ans[2];
	chunk_cell_t load[2];
	const chunk_cell_t mask = d_chunk_cell_mask[Chunk::log_elements_in_chunk+1];
	load[0]=d_chunk_p->v[in_chunk_idx];
	load[1]=d_chunk_p[1].v[in_chunk_idx];
	ans[0]=(load[0] & mask) | ((load[1] & mask)<<(Chunk::elements_in_chunk>>1));
	ans[1]=(load[1] & (mask<< (Chunk::elements_in_chunk>>1)) )|
			((load[0] >> (Chunk::elements_in_chunk>>1)) & mask);
	d_chunk_p->v[in_chunk_idx] = ans[0];
	d_chunk_p[1].v[in_chunk_idx] = ans[1];
}
__device__ void partition_general(Chunk* d_chunk_p_src, Chunk* d_chunk_p_dst,  idx_t chunk_idx, idx_t in_chunk_idx, idx_t t_dim){
	if(chunk_idx & 1)
		d_chunk_p_dst[(chunk_idx>>1) + (1<<(t_dim-1))].v[in_chunk_idx] = d_chunk_p_src[chunk_idx].v[in_chunk_idx];
	else
		d_chunk_p_dst[chunk_idx>>1].v[in_chunk_idx] = d_chunk_p_src[chunk_idx].v[in_chunk_idx];
}

__global__ void k_partition_iteration_two_chunks(Chunk* d_chunk_p_src, len_t p_len){
	idx_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	idx_t chunk_idx = (idx >> Chunk::log_cells_in_chunk)<<1;
	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	if(chunk_idx >= p_len){
		return;
	}
	partition_two_chunks(d_chunk_p_src+chunk_idx, in_chunk_idx);
}
__global__ void k_partition_iteration_in_chunk(Chunk* d_chunk_p_src, len_t p_len, len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	idx_t chunk_idx = idx >> Chunk::log_cells_in_chunk;
	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	if(chunk_idx >= p_len){
		return;
	}
	partition_in_chunk(d_chunk_p_src+chunk_idx,p_len,in_chunk_idx,t_dim);
}
__global__ void k_partition_iteration_general(Chunk* d_chunk_p_src, Chunk* d_chunk_p_dst, len_t p_len, len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	idx_t chunk_idx = idx >> Chunk::log_cells_in_chunk;
	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	idx_t base = chunk_idx ^ (chunk_idx & andMask(t_dim-Chunk::log_elements_in_chunk));
	chunk_idx &= andMask(t_dim-Chunk::log_elements_in_chunk);
	if(base+chunk_idx >= p_len)
		return;
	partition_general(d_chunk_p_src+base,d_chunk_p_dst+base, chunk_idx, in_chunk_idx,t_dim-Chunk::log_elements_in_chunk );
}
/*
 * If t_dim > Chunk::log_elements_in_chunk+1 the result is written in dst, otherwise it will be written in src.
 */
bool GPU_FFT::partition(Chunk* d_chunk_p_src, Chunk* d_chunk_p_dst, len_t p_len, len_t t_dim){
	len_t threads = p_len << Chunk::log_cells_in_chunk;
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
	k_partition_iteration_in_chunk<<<gridSize,blockSize>>>(d_chunk_p_src,p_len,t_dim);
	if(t_dim >= Chunk::log_elements_in_chunk +1){
      	dim3 gridSizeTwoChunks(sizeCiel(threads>>1,max_block_size),1,1);
		k_partition_iteration_two_chunks<<<gridSizeTwoChunks,blockSize>>>(d_chunk_p_src,p_len);
	}
	if(t_dim > Chunk::log_elements_in_chunk+1){
		k_partition_iteration_general<<<gridSize,blockSize>>>(d_chunk_p_src,d_chunk_p_dst,p_len,t_dim);
		return true;
	}
	return false;
}

//__global__ void k_xor_chunk_vector_by_single_chuk(Chunk* chunk_vec, Chunk* single_chunk, len_t chunk_vec_len){
//	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
//	idx_t chunk_idx = idx >> Chunk::log_cells_in_chunk;
//	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
//	if(chunk_idx >= chunk_vec_len){
//		return;
//	}
//	chunk_vec[chunk_idx].v[in_chunk_idx] ^= single_chunk->v[in_chunk_idx];
//	return;
//}
__global__ void k_copy_and_shift_vec(Chunk* d_chunk_src, Chunk* d_chunk_dst, len_t p_len){

	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
	__shared__ Chunk c_shared[shared_len<<1];

	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);

	if(chunk_idx  >= p_len){
		return;
	}

	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);

	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
		my_shared_chunk->v[in_chunk_idx+i]=d_chunk_src[chunk_idx].v[in_chunk_idx+i];
		my_shared_chunk[1].v[in_chunk_idx+i]=d_linear_mul->v[in_chunk_idx+i];
	}

	chunk_cell_t tmp;
	for(unsigned int i = 0 ; i < Element::ord ; i+=warp_size){
		tmp = my_shared_chunk->v[in_chunk_idx+i];
		my_shared_chunk->v[in_chunk_idx+i] = (tmp & 0xaaaaaaaa) ^ ((tmp & 0xaaaaaaaa)>>1);
		d_chunk_dst[chunk_idx].v[in_chunk_idx+i] = (tmp & 0x55555555) ^ ((tmp & 0x55555555)<<1);
	}
	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
	Chunk::chunk_reduce(d_chunk_src+chunk_idx,my_shared_chunk,in_chunk_idx);
	for(unsigned int i = 0 ; i < Element::ord ; i+=warp_size){
		d_chunk_src[chunk_idx].v[in_chunk_idx+i] ^= d_chunk_dst[chunk_idx].v[in_chunk_idx+i];
	}
	return;
}
//__global__ void k_xor_chunk_vectors(Chunk* d_chunk, Chunk* x, len_t p_len){
//	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
//	idx_t chunk_idx = idx >> Chunk::log_cells_in_chunk;
//	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
//	if(chunk_idx >= p_len){
//		return;
//	}
//	d_chunk[chunk_idx].v[in_chunk_idx] ^= x[chunk_idx].v[in_chunk_idx];
//}
void GPU_FFT::linearEvaluation(Chunk* d_chunk_p,Chunk* d_chunk_p_cpy, len_t p_len){
	len_t threads = p_len << Chunk::log_threads_in_chunk;
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
	k_copy_and_shift_vec<<<gridSize,blockSize>>>(d_chunk_p,d_chunk_p_cpy,p_len);
}
__global__ void k_subspaceAdd_general(Chunk* d_a, len_t a_len, len_t b_len){
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	idx_t chunk_idx = (idx >> Chunk::log_cells_in_chunk);
	chunk_idx = ((chunk_idx / b_len)*(2*b_len)) +b_len+(chunk_idx & (b_len-1));
	if(chunk_idx  >= a_len){
		return;
	}
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	d_a[chunk_idx].v[in_chunk_idx]^=d_a[chunk_idx-b_len].v[in_chunk_idx];
}
__global__ void k_subspaceMult_general(Chunk* d_a, Chunk* d_b , len_t b_len , len_t a_len){
	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
	__shared__ Chunk c_shared[shared_len<<1];
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	idx_t chunk_idx = (((idx >> Chunk::log_threads_in_chunk))/b_len)*(2*b_len)+b_len;
	const idx_t b_chunk_idx = (idx>>Chunk::log_threads_in_chunk) & (b_len - 1);
	chunk_idx+=b_chunk_idx;
	if(chunk_idx  >= a_len){
		return;
	}
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
		my_shared_chunk->v[in_chunk_idx+i]=d_a[chunk_idx].v[in_chunk_idx+i];
		my_shared_chunk[1].v[in_chunk_idx+i]=d_b[b_chunk_idx].v[in_chunk_idx+i];
	}
	Chunk::clmul_by_chunk_bShuffle_xor(d_a[chunk_idx],d_b[b_chunk_idx],in_chunk_idx,d_a[chunk_idx-b_len],0);
//	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
//	Chunk::chunk_reduce_xor(d_a+chunk_idx,my_shared_chunk,in_chunk_idx,d_a+chunk_idx-b_len,0);
}
__global__ void k_subspaceMult_chunk_and_add(Chunk* d_a, Chunk* d_b , len_t log_elements_in_b ,len_t a_len ){
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	if(chunk_idx  >= a_len){
		return;
	}
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	Chunk::clmul_by_chunk_bShuffle_xor_mask(d_a[chunk_idx],*d_b,in_chunk_idx,d_a[chunk_idx],1<<log_elements_in_b,log_elements_in_b);
}
/*
 * Dim = Size of U.
 */
void GPU_FFT::WFromUV(Chunk* d_chunk_p, len_t p_len, Chunk* subspace, len_t dim){
	if(dim>=Chunk::log_elements_in_chunk){
		len_t threadsMul = p_len << (Chunk::log_threads_in_chunk-1);
		len_t threadsAdd = p_len << (Chunk::log_cells_in_chunk-1);
		dim3 block(multThreadsInBlock,1,1);
		dim3 gridMul(sizeCiel(threadsMul,multThreadsInBlock),1,1);
		dim3 gridAdd(sizeCiel(threadsAdd,multThreadsInBlock),1,1);
		k_subspaceMult_general<<<gridMul,block>>>(d_chunk_p, subspace, (1<<(dim-Chunk::log_elements_in_chunk)),p_len);
		k_subspaceAdd_general<<<gridAdd,block>>>(d_chunk_p,p_len,1<<(dim-Chunk::log_elements_in_chunk));
	} else {
		len_t threadsMul = p_len << Chunk::log_threads_in_chunk;
		dim3 block(multThreadsInBlock,1,1);
		dim3 gridMul(sizeCiel(threadsMul,multThreadsInBlock),1,1);
		k_subspaceMult_chunk_and_add<<<gridMul,block>>>(d_chunk_p,subspace, dim , p_len);
	}
}
void GPU_FFT::setUpConstantMemory(const FFT* fft){
	/*
	 * Masks for taylor expansion on small dimensions.
	 */
	hipMemcpyToSymbol(HIP_SYMBOL(d_chunk_cell_mask),taylorExp_masks,sizeof(chunk_cell_t)*(Chunk::log_elements_in_chunk+2));
	/*
	 * Masks for WFromUV
	 */
	hipMemcpyToSymbol(HIP_SYMBOL(d_alter_mask),alter_masks,sizeof(chunk_cell_t)*(Chunk::log_elements_in_chunk+1));
	/*
	 * Table for partition operation on small dimension.
	 */
	hipMemcpyToSymbol(HIP_SYMBOL(d_partition),partition_byte,sizeof(chunk_cell_t)*(1<<bits_in_byte));
	/*
	 * Chunks to multiply and add when calculating linear functions.
	 */
	hipMemcpyToSymbol(HIP_SYMBOL(d_linear_mul),&fft->linear_mul,sizeof(Chunk));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ilinear_mul),&fft->ilinear_mul,sizeof(Chunk));
}
/*
 * p is a pointer to a chunk array on gpu, l is its length.
 * prints that array.
 */
void printChunkOnGPU(Chunk* p , len_t l){
// TODO: No real need for printing and this break the building of FFT as separate library
/*
	Chunk* h_p = Tests::copyFromGPU(p,l);
	for(unsigned int i = 0 ; i < l ; ++i){
		h_p[i].print();
		std::cout << std::endl << std::endl;

	}
	free(h_p);
	std::cout << std::endl << std::endl;
*/
}

/*
 * This is the gpu implementation of the FFT when it fits into a single thread block.
 * In that case, the whole FFT can be calculated using a single kernel invocation and some additional
 * synchronization primitives.
 *
 * In this implementation, each WARP is responsible for a single chunk, therefore if 32 threads compose a single warp and
 * we use 1024 threads in a thread block, then each thread block is responsible for 32 chunks.
 *
 * 0) First we load ALL subspaces and exponents into global memory in a special pre-allocated array.
 * We also allocate 64 Chunks long array in shared memory.
 *
 * 1) We load the shared memory with the relevant polynomial (depends on the index).
 * 2) Series of Multiexp -> taylorExp -> partition, all inside local memory.
 * 	2.1) Before Each Multiexp we load 32 chunks of exps to shared memory.
 * 3) Linear evaluation.
 * 4) Series of WFromUV, while loading subspaces from global memory.
 *
 * This whole implementation assumes that:
 * 		Chunk::log_cells_in_chunk = 1+ Chunk::log_elements_in_chunk
 */

__device__ void InTB_preMulExpand(	Chunk* const d_a,
									const Chunk* const d_b,
									const len_t b_len ,
									const idx_t idx ,
									const idx_t chunkIdx ,
									const idx_t in_chunk_idx){
	chunk_cell_t load[2];
	load[0]=d_a[chunkIdx].v[in_chunk_idx];
	load[1]=d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];
	__syncthreads();
	d_a[chunkIdx<<1].v[in_chunk_idx] = load[0];
	d_a[chunkIdx<<1].v[in_chunk_idx + Chunk::elements_in_chunk] = load[1];
	d_a[(chunkIdx<<1)+1].v[in_chunk_idx]
	                       = d_b[chunkIdx & (b_len-1)].v[in_chunk_idx];
	d_a[(chunkIdx<<1)+1].v[in_chunk_idx + Chunk::elements_in_chunk]
	                       = d_b[chunkIdx & (b_len-1)].v[in_chunk_idx + Chunk::elements_in_chunk];
	__syncthreads();
}
/*
 * Takes every second chunk and puts all of them in the first half.
 * Second half output is unknown.
 * 1) d_a - Pointer for shmem array of the polynomial.
 * 2) idx - index of the thread.
 * 3) chunkIdx - (idx / Chunk::elemetns_in_chunk).
 * 4) in_chunk_idx - (idx % Chunk::elements_in_chunk).
 */
__device__ void InTB_postMulShrink(	Chunk * const d_a,
									const idx_t idx ,
									const idx_t chunkIdx ,
									const idx_t in_chunk_idx){
	chunk_cell_t load[2];
	load[0]=d_a[chunkIdx<<1].v[in_chunk_idx];
	load[1]=d_a[chunkIdx<<1].v[in_chunk_idx+Chunk::elements_in_chunk];
	__syncthreads();
	d_a[chunkIdx].v[in_chunk_idx]=load[0];
	d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk]=load[1];
	__syncthreads();
}
/*
 * Multiplies d_a[0] by d_a[1] and writes the answer at d_a[0].
 * The state of d_a[1] is unknown at the end.
 *
 * 1)	d_a - the chunk the mult.
 * 2) 	in_chunk_idx - Is thread index % Chunk::elements_in_chunk
 */
__device__ void InTB_Mult(Chunk* const d_a, const idx_t in_chunk_idx){
	Chunk::clmul_by_chunk(*d_a,d_a[1],in_chunk_idx,d_a);
	Chunk::chunk_reduce(d_a,d_a,in_chunk_idx);
	__syncthreads();
}
/*
 * This is the multi exponentiation.
 * 1) d_a - pointer to the shmem.
 * 2) exp - pointer to the exponents to multiply (global mem).
 * 3) dim - dimension of multiexponentiation.
 * 4) idx - thread index.
 * 5) chunkIdx - idx/Chunk::elements_in_chunk.
 * 6) in_chunk_idx - idx%Chunk::elements_in_chunk.
 */
 __device__ void InTB_MultiExp(	Chunk* const d_a,
		 	 	 	 	 	 	const Chunk* const exp,
		 	 	 	 	 	 	const len_t exp_len,
		 	 	 	 	 	 	const idx_t idx,
		 	 	 	 	 	 	const idx_t chunkIdx,
		 	 	 	 	 	 	const idx_t in_chunk_idx,
		 	 	 	 	 	 	const len_t half_d_a_len){
	chunk_cell_t load[2];
	chunk_cell_t swap;
	load[0] = d_a[half_d_a_len + chunkIdx].v[in_chunk_idx];
	load[1] = d_a[half_d_a_len + chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];

	InTB_preMulExpand(d_a,exp,exp_len,idx,chunkIdx,in_chunk_idx);
	InTB_Mult(d_a+(chunkIdx<<1),in_chunk_idx);
	InTB_postMulShrink(d_a,idx,chunkIdx,in_chunk_idx);

	swap = load[0];
	load[0]=d_a[chunkIdx].v[in_chunk_idx];
	d_a[chunkIdx].v[in_chunk_idx] = swap;
	swap = load[1];
	load[1]=d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] = swap;

	InTB_preMulExpand(d_a,exp+((half_d_a_len) & (exp_len>>1)),exp_len,idx,chunkIdx,in_chunk_idx);
	InTB_Mult(d_a+(chunkIdx<<1),in_chunk_idx);
	InTB_postMulShrink(d_a,idx,chunkIdx,in_chunk_idx);

	d_a[half_d_a_len + chunkIdx].v[in_chunk_idx] = d_a[chunkIdx].v[in_chunk_idx];
	d_a[half_d_a_len + chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] =
			d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk];
	d_a[chunkIdx].v[in_chunk_idx] = load[0];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] = load[1];
	__syncthreads();

 }
/*
 * Taylor Expansion
 *
 * 1) d_a is the pointer to the beginning of the polynomial in shmem.
 * 2) dim>=2. Also dim<=10.
 * 3) idx - the idx of the thread.
 * 4) in_chunk_idx - (idx % (Chunk::chunk_cells_in_chunk)).
 * 5) chunkIdx -  index of chunk INSIDE sub-polynomial.
 */
__device__ void InTB_TaylorExp(	Chunk* const d_a,
								idx_t dim,
								const idx_t idx,
								const idx_t in_chunk_idx,
								const idx_t chunkIdx,
								const idx_t half_d_a_len){
	len_t l,s;
	/*
	 * If more than a single chunk is needed for a subpolynomial
	 */
	if(dim> Chunk::log_elements_in_chunk){
		for(; dim > Chunk::log_elements_in_chunk+1 ; --dim){
				l=dim-Chunk::log_elements_in_chunk;
				s=1<<(l-2);
				l=((chunkIdx>>(l-2))<<l)+(chunkIdx&andMask(l-2));
				d_a[l+(2*s)].v[in_chunk_idx]^=d_a[l+(3*s)].v[in_chunk_idx];
				d_a[l+(s)].v[in_chunk_idx]^=d_a[l+(2*s)].v[in_chunk_idx];
			__syncthreads();
		}
		/*
		 * Subpolynomial of size 2 chunks
		 */
		s=Chunk::cells_in_chunk>>1;
		l=chunkIdx*2;
		for(unsigned int i = 0 ; i < 2 ; ++i){
			d_a[l+1+i*half_d_a_len].v[in_chunk_idx]^=
					(d_a[l+1+i*half_d_a_len].v[in_chunk_idx]>>(Chunk::elements_in_chunk>>1));
			d_a[l+i*half_d_a_len].v[in_chunk_idx]^=
					(d_a[l+1+i*half_d_a_len].v[in_chunk_idx]<<(Chunk::elements_in_chunk>>1));
		}
		--dim;
		__syncthreads();
	}
	unsigned int i = dim;
	/*
	 * Code duplicated to prevent additional register usage for loop counter, assuming that:
	 * 		Chunk::log_cells_in_chunk = Chunk::log_elements_in_chunk+1.
	 */
	for(unsigned int j = 0 ; j < 4 ; ++j ) {
		chunk_cell_t cell = d_a[chunkIdx+j*(half_d_a_len>>1)].v[in_chunk_idx];
		for(; dim >=2 ; --dim){
			cell ^= (cell &
					(d_chunk_cell_mask[dim]<<((1<<dim)-(1<<(dim-2)))))>>(1<<(dim-2));
			cell ^= (cell &
					(d_chunk_cell_mask[dim]<<(1<<(dim-1))))>>(1<<(dim-2));
		}
		d_a[chunkIdx+j*(half_d_a_len>>1)].v[in_chunk_idx] = cell ;
		dim = i;
	}
	__syncthreads();
}
/*
 * This is the partition operation
 * 1)d_a - the shmem ptr for the polynomial.
 * 2) dim - the dimension of partition.
 * 3) idx - the thread index.
 * 4) chunk_idx - (idx >> Chunk::log_elements_in_chunk).
 * 5) in_chunk_idx - (idx % Chunk::elements_in_chunk).
 * 6) d_a_len - log  length of subpoly in chunks.
 */
__device__ void InTB_partition(	Chunk* const d_a,
								const len_t dim,
								const idx_t idx,
								const idx_t chunk_idx,
								const idx_t in_chunk_idx,
								const len_t d_a_len,
								const len_t half_d_a_len){
	/*
	 * Assumes:
	 *     Chunk::log_cells_in_chunk = 1+ Chunk::log_elements_in_chunk
	 */
	idx_t i=2;
	chunk_cell_t ans[2];
	for(unsigned int j = 0 ; j < 2 ; ++j){
		ans[0]= d_a[chunk_idx + j*half_d_a_len].v[in_chunk_idx];
		ans[1]=d_a[chunk_idx + j*half_d_a_len].v[in_chunk_idx + (Chunk::cells_in_chunk>>1)];
		for(i=2; i <= Chunk::log_elements_in_chunk && i<=dim ; ++i){
			ans[0] = 	(ans[0] & d_chunk_cell_mask[i]) |
					(ans[0] & (d_chunk_cell_mask[i]<<((1<<(i))-(1<<(i-2))))) |
					(ans[0] & (d_chunk_cell_mask[i]<<(1<<(i-1))))>>(1<<(i-2)) |
					(ans[0] & (d_chunk_cell_mask[i]<<(1<<(i-2))))<<(1<<(i-2));
			ans[1] = 	(ans[1] & d_chunk_cell_mask[i]) |
					(ans[1] & (d_chunk_cell_mask[i]<<((1<<(i))-(1<<(i-2))))) |
					(ans[1] & (d_chunk_cell_mask[i]<<(1<<(i-1))))>>(1<<(i-2)) |
					(ans[1] & (d_chunk_cell_mask[i]<<(1<<(i-2))))<<(1<<(i-2));
		}
		d_a[chunk_idx + j*half_d_a_len].v[in_chunk_idx]=ans[0];
		d_a[chunk_idx + j*half_d_a_len].v[in_chunk_idx+(Chunk::cells_in_chunk>>1)] = ans[1];
	}
	__syncthreads();

	if(dim<=Chunk::log_elements_in_chunk){
		return;
	}
	idx_t s = (idx& andMask(log_max_block_size))>> Chunk::log_cells_in_chunk;
	s<<=1;
	idx_t t = idx & andMask(Chunk::log_cells_in_chunk);

	for(unsigned int j = 0 ; j < 2 ; ++j){
		ans[0]=d_a[s + j*half_d_a_len].v[t];
		ans[1]=d_a[s+1 + j*half_d_a_len].v[t];
		d_a[s+j*half_d_a_len].v[t] = (ans[0] & d_chunk_cell_mask[Chunk::log_elements_in_chunk+1]) | ((ans[1] & d_chunk_cell_mask[Chunk::log_elements_in_chunk+1])<<(Chunk::elements_in_chunk>>1));
		d_a[s+1 + j*half_d_a_len].v[t] = (ans[1] & (d_chunk_cell_mask[Chunk::log_elements_in_chunk+1]<< (Chunk::elements_in_chunk>>1)) )|
				((ans[0] >> (Chunk::elements_in_chunk>>1)) & d_chunk_cell_mask[Chunk::log_elements_in_chunk+1]);
	}
	__syncthreads();

	if(dim<=Chunk::log_elements_in_chunk+1){
		return;
	}
	ans[0] = d_a[chunk_idx].v[in_chunk_idx];
	ans[1] = d_a[chunk_idx].v[in_chunk_idx+Chunk::elements_in_chunk];
	chunk_cell_t ans_u[2];
	ans_u[0] = d_a[chunk_idx+half_d_a_len].v[in_chunk_idx];
	ans_u[1] = d_a[chunk_idx+ half_d_a_len].v[in_chunk_idx+Chunk::elements_in_chunk];
	__syncthreads();
	s = (chunk_idx >> d_a_len)<<(d_a_len);
	t = chunk_idx & andMask(d_a_len);
	if(t & 1){
		d_a[s+(t>>1) + (1<<(d_a_len-1))].v[in_chunk_idx] = ans[0];
		d_a[s+(t>>1) + (1<<(d_a_len-1))].v[in_chunk_idx+Chunk::elements_in_chunk] = ans[1];
	} else {
		d_a[s+(t>>1)].v[in_chunk_idx] = ans[0];
		d_a[s+(t>>1)].v[in_chunk_idx+Chunk::elements_in_chunk] = ans[1];
	}

	s=((chunk_idx+half_d_a_len) >> d_a_len)<<d_a_len;
	t=(chunk_idx+half_d_a_len)&andMask(d_a_len);
	if(t & 1){
		d_a[s+(t>>1) + (1<<(d_a_len-1))].v[in_chunk_idx] = ans_u[0];
		d_a[s+(t>>1) + (1<<(d_a_len-1))].v[in_chunk_idx+Chunk::elements_in_chunk] = ans_u[1];
	} else {
		d_a[s+(t>>1)].v[in_chunk_idx] = ans_u[0];
		d_a[s+(t>>1)].v[in_chunk_idx+Chunk::elements_in_chunk] = ans_u[1];
	}
	__syncthreads();

}

/*
 * This is the linear evaluation phase
 */
__device__ void InTB_LinearEvaluation(	Chunk* const d_a,
										const idx_t idx ,
										const idx_t chunkIdx ,
										const idx_t in_chunk_idx,
										const len_t half_d_a_len){
	chunk_cell_t load[4];
	load[0] = d_a[chunkIdx].v[in_chunk_idx];
	load[1] = d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];
	load[2] = d_a[half_d_a_len+chunkIdx].v[in_chunk_idx];
	load[3] = d_a[half_d_a_len+chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];

	DUP_ODD_BITS(load[0]);
	DUP_ODD_BITS(load[1]);

	DUP_EVEN_BITS(d_a[chunkIdx].v[in_chunk_idx]);
	DUP_EVEN_BITS(d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk]);
	__syncthreads();
	InTB_preMulExpand(d_a,d_linear_mul,1,idx,chunkIdx,in_chunk_idx);
	InTB_Mult(d_a+(chunkIdx<<1),in_chunk_idx);
	InTB_postMulShrink(d_a,idx,chunkIdx,in_chunk_idx);

	d_a[chunkIdx].v[in_chunk_idx] ^= load[0];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk]  ^= load[1];
	__syncthreads();

	load[0]=load[2];
	load[2]=d_a[chunkIdx].v[in_chunk_idx];
	d_a[chunkIdx].v[in_chunk_idx] = load[0];

	load[1]=load[3];
	load[3]=d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];
	d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk] = load[1];

	DUP_ODD_BITS(load[0]);
	DUP_ODD_BITS(load[1]);

	DUP_EVEN_BITS(d_a[chunkIdx].v[in_chunk_idx]);
	DUP_EVEN_BITS(d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk]);
	__syncthreads();
	InTB_preMulExpand(d_a,d_linear_mul,1,idx,chunkIdx,in_chunk_idx);
	InTB_Mult(d_a+(chunkIdx<<1),in_chunk_idx);
	InTB_postMulShrink(d_a,idx,chunkIdx,in_chunk_idx);

	d_a[chunkIdx].v[in_chunk_idx] ^= load[0];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk]  ^= load[1];
	__syncthreads();

	d_a[chunkIdx + half_d_a_len].v[in_chunk_idx] = d_a[chunkIdx].v[in_chunk_idx];
	d_a[chunkIdx + half_d_a_len].v[in_chunk_idx + Chunk::elements_in_chunk] =
			d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk];

	d_a[chunkIdx].v[in_chunk_idx]=load[2];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk]=load[3];

	__syncthreads();
}

/*
 * This is the WFromUV operation in chunk (i.e. dim<5)
 * dim = dim of subspace.
 */
__device__ void InTB_WFromUV_inChunk(	Chunk* const d_a,
										const Chunk * const subspace,
										const len_t dim,
										const idx_t idx,
										const idx_t chunkIdx,
										const idx_t in_chunk_idx,
										const len_t half_d_a_len){
	/*
	 * 1)Backup the WHOLE cell.
	 * 2)Multiply by subspace chunk (will nullify lower top).
	 * 3)Xor by itself when shifting right subspace-times.
	 * 4)Xor the load.
	 */
	chunk_cell_t load[4];
	/*
	 * Just backing up upper half
	 */
	load[2] = d_a[chunkIdx + half_d_a_len].v[in_chunk_idx];
	load[3] = d_a[chunkIdx + half_d_a_len].v[in_chunk_idx + Chunk::elements_in_chunk];

	load[0] = d_a[chunkIdx].v[in_chunk_idx] ;
	load[1] = d_a[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk] ;
	load[0] ^= ((load[0]&(d_alter_mask[dim+1]))<<(1<<dim));
	load[1] ^= ((load[1]&(d_alter_mask[dim+1]))<<(1<<dim));

	__syncthreads();

	InTB_preMulExpand(d_a,subspace,1,idx,chunkIdx,in_chunk_idx);
	InTB_Mult(d_a+(chunkIdx<<1),in_chunk_idx);
	InTB_postMulShrink(d_a,idx,chunkIdx,in_chunk_idx);
	d_a[chunkIdx].v[in_chunk_idx] ^= (load[0]^(d_a[chunkIdx].v[in_chunk_idx]>>(1<<dim)));
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] ^= (load[1]^(d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk]>>(1<<dim)));

	__syncthreads();

	/*
	 * Switching from calculating lower half , into upper half.
	 */
	load[0] = load[2];
	load[2] = d_a[chunkIdx].v[in_chunk_idx];
	d_a[chunkIdx].v[in_chunk_idx] = load[0];

	load[1] = load[3];
	load[3] = d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] = load[1];

	load[0] ^= ((load[0]&(d_alter_mask[dim+1]))<<(1<<dim));
	load[1] ^= ((load[1]&(d_alter_mask[dim+1]))<<(1<<dim));

	__syncthreads();

	InTB_preMulExpand(d_a,subspace,1,idx,chunkIdx,in_chunk_idx);
	InTB_Mult(d_a+(chunkIdx<<1),in_chunk_idx);
	InTB_postMulShrink(d_a,idx,chunkIdx,in_chunk_idx);
	d_a[chunkIdx].v[in_chunk_idx] ^= (load[0]^(d_a[chunkIdx].v[in_chunk_idx]>>(1<<dim)));
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] ^= (load[1]^(d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk]>>(1<<dim)));

	__syncthreads();

	d_a[chunkIdx + half_d_a_len].v[in_chunk_idx] = d_a[chunkIdx].v[in_chunk_idx];
	d_a[chunkIdx + half_d_a_len].v[in_chunk_idx + Chunk::elements_in_chunk] =
			d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk];

	d_a[chunkIdx].v[in_chunk_idx] = load[2];
	d_a[chunkIdx].v[in_chunk_idx + Chunk::elements_in_chunk] = load[3];

	__syncthreads();
}
/*
 * This is when the SUBSPACE dim it >=5.
 * 1) d_a - shmem ptr.
 * 2) subspace - global mem ptr of length (1<<(dim-1-Chunk::log_elements_in_chunk)) to a subspace of dim (dim-1).
 * 3) dim - dimension of subspace.
 * 4) idx - threadIdx.
 */
__device__ void InTB_WFromUV_outChunk(	Chunk* const d_a,
										const Chunk* const subspace,
										const len_t dim,
										const idx_t idx,
										const len_t half_d_a_len){
	/*
	 * 1) Backup everything on register.
	 * 2) Multiply by subspace only bottom halves of the sub-polynomials.
	 */
	idx_t chunkIdx64[2];
	idx_t original_idx[2];
	original_idx[0] = ((idx& andMask(log_max_block_size)) >> Chunk::log_cells_in_chunk);
	original_idx[1] = original_idx[0]+(max_block_size>>Chunk::log_cells_in_chunk);
	chunkIdx64[0] = (((original_idx[0])>>(dim-Chunk::log_elements_in_chunk))<<(dim+1-Chunk::log_elements_in_chunk)) +
			(original_idx[0] & andMask(dim-Chunk::log_elements_in_chunk));
	chunkIdx64[1] = (((original_idx[1])>>(dim-Chunk::log_elements_in_chunk))<<(dim+1-Chunk::log_elements_in_chunk)) +
			(original_idx[1] & andMask(dim-Chunk::log_elements_in_chunk));
	idx_t in_chunk_idx64 = (idx) & andMask(Chunk::log_cells_in_chunk);
	chunk_cell_t load[4];
	//Backing up everything on local memory.
	load[0]= d_a[chunkIdx64[0]].v[in_chunk_idx64];
	load[1]= d_a[chunkIdx64[0] + (1<<(dim-Chunk::log_elements_in_chunk))].v[in_chunk_idx64];
	load[2] = d_a[chunkIdx64[1]].v[in_chunk_idx64];
	load[3] = d_a[chunkIdx64[1]+ (1<<(dim-Chunk::log_elements_in_chunk))].v[in_chunk_idx64];

	__syncthreads();

	//Preparing data for multiplication.
	d_a[(original_idx[0])<<1].v[in_chunk_idx64]=load[1];
	d_a[(original_idx[1])<<1].v[in_chunk_idx64]=load[3];

	//If my chunk should be multiplied, I load the relevant subspace chunk to the next chunk, so they will be multiplied.
	//My chunk should be multiplied if my chunkIdx/(sizeOfSubspace in Chunk) is odd.
	//chunkIdx is the same for all threads in the warp - so no divergence is possible.
	d_a[(original_idx[0]<<1)+1].v[in_chunk_idx64]=
		subspace[original_idx[0] & andMask(dim-Chunk::log_elements_in_chunk)].v[in_chunk_idx64];
	d_a[(original_idx[1]<<1)+1].v[in_chunk_idx64]=
		subspace[original_idx[1] & andMask(dim-Chunk::log_elements_in_chunk)].v[in_chunk_idx64];
	__syncthreads();
	InTB_Mult(d_a+(((idx&andMask(log_max_block_size))>>Chunk::log_threads_in_chunk)<<1),idx&andMask(Chunk::log_threads_in_chunk));

	chunk_cell_t t[2];
	t[0]= d_a[original_idx[0]<<1].v[in_chunk_idx64];
	t[1]= d_a[original_idx[1]<<1].v[in_chunk_idx64];
	__syncthreads();
	load[1]^=load[0];
	load[3]^=load[2];
	d_a[chunkIdx64[0]+(1<<(dim-Chunk::log_elements_in_chunk))].v[in_chunk_idx64] = t[0] ^ load[1];
	d_a[chunkIdx64[0]].v[in_chunk_idx64] = t[0] ^ load[0];
	d_a[chunkIdx64[1]+(1<<(dim-Chunk::log_elements_in_chunk))].v[in_chunk_idx64] = t[1] ^ load[3];
	d_a[chunkIdx64[1]].v[in_chunk_idx64] = t[1] ^ load[2];
	__syncthreads();
}

__global__ void
//__launch_bounds__(1<<log_max_block_size)
k_gpuFFT_InTB(Chunk* const d_a, Chunk** subspaces, Chunk** exps, len_t dim, const len_t poly_len){
	const len_t half_d_a_len = (max_block_size>>Chunk::log_threads_in_chunk);
	__shared__ Chunk s[half_d_a_len<<1];
	const idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	const idx_t poly_idx = (idx >> log_max_block_size)<<(log_max_block_size+1-Chunk::log_elements_in_chunk);
	const idx_t chunkIdx = (idx  & andMask(log_max_block_size))>> Chunk::log_elements_in_chunk;
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_elements_in_chunk);
	if(poly_idx+chunkIdx < poly_len){
		s[chunkIdx].v[in_chunk_idx] = d_a[(poly_idx) + chunkIdx].v[in_chunk_idx];
		s[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk] = d_a[(poly_idx) + chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];
	}
	if(poly_idx+chunkIdx+half_d_a_len < poly_len){
		s[chunkIdx+half_d_a_len].v[in_chunk_idx] = d_a[(poly_idx) + chunkIdx + half_d_a_len].v[in_chunk_idx];
		s[chunkIdx+half_d_a_len].v[in_chunk_idx+Chunk::elements_in_chunk] =
				d_a[(poly_idx) + chunkIdx + half_d_a_len].v[in_chunk_idx+Chunk::elements_in_chunk];
	}

	__syncthreads();
	idx_t i = dim;
	for(; i > Chunk::log_elements_in_chunk ; --i){
		InTB_MultiExp(s,exps[i],1<<(i-Chunk::log_elements_in_chunk),idx,chunkIdx,in_chunk_idx,half_d_a_len);
		//The problem is in TaylorExp - s is nullified!
		InTB_TaylorExp(s,i,idx,idx & andMask(Chunk::log_cells_in_chunk),(idx&andMask(log_max_block_size)) >> Chunk::log_cells_in_chunk,half_d_a_len);
		InTB_partition(s,i,idx,chunkIdx,in_chunk_idx,(i-Chunk::log_elements_in_chunk),half_d_a_len);
	}
	for(; i >= 2; --i){
		InTB_MultiExp(s,exps[i],1,idx,chunkIdx,in_chunk_idx,half_d_a_len);
		InTB_TaylorExp(s,i,idx,idx & andMask(Chunk::log_cells_in_chunk),(idx&andMask(log_max_block_size)) >> Chunk::log_cells_in_chunk,half_d_a_len);
		InTB_partition(s,i,idx,chunkIdx,in_chunk_idx,0,half_d_a_len);
	}
	InTB_LinearEvaluation(s,idx,chunkIdx,in_chunk_idx,half_d_a_len);
	for(; i < Chunk::log_elements_in_chunk && i<dim ; ++i){
		InTB_WFromUV_inChunk(s,subspaces[i],i,idx,chunkIdx,in_chunk_idx,half_d_a_len);
	}
	for( ; i < dim ; ++i){
		InTB_WFromUV_outChunk(s,subspaces[i],i,idx,half_d_a_len);
	}
	if(poly_idx+chunkIdx < poly_len){
		d_a[(poly_idx) + chunkIdx].v[in_chunk_idx] = s[chunkIdx].v[in_chunk_idx];
		d_a[(poly_idx) + chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk] = s[chunkIdx].v[in_chunk_idx+Chunk::elements_in_chunk];
	}
	if(poly_idx+chunkIdx+half_d_a_len < poly_len){
		d_a[(poly_idx) + chunkIdx + half_d_a_len].v[in_chunk_idx] = s[chunkIdx+half_d_a_len].v[in_chunk_idx];
		d_a[(poly_idx) + chunkIdx + half_d_a_len].v[in_chunk_idx+Chunk::elements_in_chunk] =
				s[chunkIdx+half_d_a_len].v[in_chunk_idx+Chunk::elements_in_chunk];
	}
}
 void GPU_FFT::fft_gpu_InTB(len_t dim, const FFT* const fft, Chunk* const p, len_t p_len){
	 len_t threads = p_len<<Chunk::log_elements_in_chunk;
	 dim3 block(max_block_size,1,1);
	 dim3 grid(sizeCiel(threads,max_block_size),1,1);
	 Chunk* d_subspaces[log_max_block_size+2];
	 Chunk* d_exps[log_max_block_size+2];
	 Chunk** d_subspaces_allocated;
	 Chunk** d_exps_allocated;

	 hipMalloc(&d_subspaces_allocated,sizeof(Chunk*)*(log_max_block_size+2));
	 hipMalloc(&d_exps_allocated,sizeof(Chunk*)*(log_max_block_size+2));

	 len_t fftSize = fft->basis.getSize();
	 for(unsigned int i = 2 ; i <= dim && i<=Chunk::log_elements_in_chunk ; ++i){
		 hipMalloc(&(d_subspaces[i-1]),sizeof(Chunk));
		 hipMalloc(&(d_exps[i]),sizeof(Chunk));
		 hipMemcpy(d_subspaces[i-1],fft->gpu_subspace[fftSize-i],sizeof(Chunk),hipMemcpyHostToDevice);
		 hipMemcpy(d_exps[i],fft->gpu_exp[fftSize-i],sizeof(Chunk),hipMemcpyHostToDevice);
	 }
	 for(unsigned int i = Chunk::log_elements_in_chunk+1 ; i <= dim ; ++i){
		 hipMalloc(&(d_subspaces[i-1]),sizeof(Chunk)*(1<<(i-1-Chunk::log_elements_in_chunk)));
		 hipMalloc(&(d_exps[i]),sizeof(Chunk)*(1<<(i-Chunk::log_elements_in_chunk)));
		 hipMemcpy(d_subspaces[i-1],fft->gpu_subspace[fftSize-i],sizeof(Chunk)*(1<<(i-1-Chunk::log_elements_in_chunk)),hipMemcpyHostToDevice);
		 hipMemcpy(d_exps[i],fft->gpu_exp[fftSize-i],sizeof(Chunk)*(1<<(i-Chunk::log_elements_in_chunk)),hipMemcpyHostToDevice);
	 }

	 hipMemcpy(d_subspaces_allocated,d_subspaces,sizeof(Chunk*)*(log_max_block_size+2),hipMemcpyHostToDevice);
	 hipMemcpy(d_exps_allocated,d_exps,sizeof(Chunk*)*(log_max_block_size+2),hipMemcpyHostToDevice);

	 k_gpuFFT_InTB<<<grid,block>>>(p,d_subspaces_allocated,d_exps_allocated,dim,p_len);
	 for(unsigned int i = 2 ; i<= dim ; ++i){
		 hipFree(d_subspaces[i-1]);
		 hipFree(d_exps[i]);
	 }

	 hipFree(d_subspaces_allocated);
	 hipFree(d_exps_allocated);
 }
 void GPU_FFT::fft_gpu(const FFT* const fft,Polynomial* P){
	 	 Chunk::setMod();
		setUpConstantMemory(fft);
		Element* normalized_P = *P;
		len_t p_len = 1<<fft->basis.getSize();
		len_t fixed_len = MAX((p_len),warp_size);
		if(p_len < fixed_len) {
			normalized_P = (Element*)malloc(sizeof(Element)*fixed_len);
			memset(normalized_P,0,sizeof(Element)*fixed_len);
			memcpy(normalized_P,*P,sizeof(Element)*p_len);
		}
		fixed_len >>= Chunk::log_elements_in_chunk;
		p_len = fixed_len;
		Elements_Chunk* d_p;
		Chunk* d_chunk_P;
		Chunk* d_chunk_P_cpy;
		Chunk* d_swap;
		hipMalloc(&d_p,sizeof(Elements_Chunk)*(fixed_len));
		hipMalloc(&d_chunk_P,sizeof(Chunk)*(fixed_len));
		hipMemcpy(d_p,normalized_P,sizeof(Elements_Chunk)*(fixed_len),hipMemcpyHostToDevice);
		Chunk::normalToChunk((Elements_Chunk*)d_p,d_chunk_P,fixed_len,false);
		hipFree(d_p);
		hipMalloc(&d_chunk_P_cpy,sizeof(Chunk)*(fixed_len));
		len_t dim = fft->basis.getSize();
#ifdef __MEASURE
#ifdef __GNUC__
		timespec start,end;
		clock_gettime(CLOCK_REALTIME,&start);
#endif	//#ifdef __GNUC__
#endif	//#ifdef __MEASURE
		/*
		 * The Algorithm
		 */
		unsigned int i = dim;
		for(; i > 1 ; --i){
			multiExponentiate_gpu(fft,d_chunk_P,p_len,i,d_chunk_P_cpy);
			taylorExpansion_gpu(fft,d_chunk_P,p_len,i);
			if(partition(d_chunk_P,d_chunk_P_cpy,p_len,i)){
				d_swap = d_chunk_P;
				d_chunk_P=d_chunk_P_cpy;
				d_chunk_P_cpy = d_swap;
			}
		}
//		fft_gpu_InTB(i,fft,d_chunk_P,p_len);
		linearEvaluation(d_chunk_P,d_chunk_P_cpy,p_len);
		for(; i < dim ; ++i ){
			hipMemcpy(d_chunk_P_cpy,fft->gpu_subspace[dim-1-i],sizeof(Chunk)<<(MAX((int)i-(int)Chunk::log_elements_in_chunk,0)),hipMemcpyHostToDevice);
			WFromUV(d_chunk_P,p_len,d_chunk_P_cpy,i);
		}
		/*
		 * End of algorithm
		 */
#ifdef __MEASURE
#ifdef __GNUC__
		clock_gettime(CLOCK_REALTIME,&end);
		double elapsed_secs = end.tv_sec - start.tv_sec
				+ ((double) (end.tv_nsec - start.tv_nsec)) / ((double) 1000000000);
		std::cout << elapsed_secs << "GpU!!" <<std::endl;
#endif	//#ifdef __GNUC__
#endif	//#ifdef __MEASURE

		hipFree(d_chunk_P_cpy);
		hipMalloc(&d_p,sizeof(Elements_Chunk)*(fixed_len));
		Chunk::chunkToNormal(d_chunk_P,(Elements_Chunk*)d_p,fixed_len,false);
		hipMemcpy(normalized_P,d_p,sizeof(Elements_Chunk)*(fixed_len),hipMemcpyDeviceToHost);
		hipFree(d_chunk_P);
		hipFree(d_p);
		if(normalized_P != *P){
			free(*P);
			*P=normalized_P;
		}
	 }
/*
 * This is the inverse FFT implementation
 */

 void GPU_FFT::ifft_gpu(const FFT* const fft,Polynomial* P){
	 	 Chunk::setMod();
		setUpConstantMemory(fft);
		Element* normalized_P = *P;
		len_t p_len = 1<<fft->basis.getSize();
		len_t fixed_len = MAX((p_len),warp_size);
		if(p_len < fixed_len) {
			normalized_P = (Element*)malloc(sizeof(Element)*fixed_len);
			memset(normalized_P,0,sizeof(Element)*fixed_len);
			memcpy(normalized_P,*P,sizeof(Element)*p_len);
		}
		fixed_len >>= Chunk::log_elements_in_chunk;
		p_len = fixed_len;
		Elements_Chunk* d_p;
		Chunk* d_chunk_P;
		Chunk* d_chunk_P_cpy;
		Chunk* d_swap;
		hipMalloc(&d_p,sizeof(Elements_Chunk)*(fixed_len));
		hipMalloc(&d_chunk_P,sizeof(Chunk)*(fixed_len));
		hipMemcpy(d_p,normalized_P,sizeof(Elements_Chunk)*(fixed_len),hipMemcpyHostToDevice);
		Chunk::normalToChunk((Elements_Chunk*)d_p,d_chunk_P,fixed_len,false);
		hipFree(d_p);
		hipMalloc(&d_chunk_P_cpy,sizeof(Chunk)*(fixed_len));
		len_t dim = fft->basis.getSize();

#ifdef __MEASURE
#ifdef __GNUC__
		timespec start,end;
		clock_gettime(CLOCK_REALTIME,&start);
#endif	//#ifdef __GNUC__
#endif  //#ifdef __MEASURE
		/*
		 * The Algorithm
		 */
		unsigned int i = dim-1;
		for(; i >= 1 ; --i ){
			hipMemcpy(d_chunk_P_cpy,fft->gpu_subspace[dim-1-i],sizeof(Chunk)<<(MAX((int)i-(int)Chunk::log_elements_in_chunk,0)),hipMemcpyHostToDevice);
			UVFromW(d_chunk_P,p_len,d_chunk_P_cpy,i);
		}
		ilinearEvaluation(d_chunk_P,d_chunk_P_cpy,p_len);
		for(i=2; i <= dim ; ++i){
			if(ipartition(d_chunk_P,d_chunk_P_cpy,p_len,i)){
				d_swap = d_chunk_P;
				d_chunk_P=d_chunk_P_cpy;
				d_chunk_P_cpy = d_swap;
			}
			itaylorExpansion_gpu(fft,d_chunk_P,p_len,i);
			imultiExponentiate_gpu(fft,d_chunk_P,p_len,i,d_chunk_P_cpy);
		}
		/*
		 * End of algorithm
		 */
#ifdef __MEASURE
#ifdef __GNUC__
		clock_gettime(CLOCK_REALTIME,&end);
		double elapsed_secs = end.tv_sec - start.tv_sec
				+ ((double) (end.tv_nsec - start.tv_nsec)) / ((double) 1000000000);
		std::cout << elapsed_secs << "GpU!!" <<std::endl;
#endif	//#ifdef __GNUC__
#endif  //#ifdef __MEASURE

		hipFree(d_chunk_P_cpy);
		hipMalloc(&d_p,sizeof(Elements_Chunk)*(fixed_len));
		Chunk::chunkToNormal(d_chunk_P,(Elements_Chunk*)d_p,fixed_len,false);
		hipMemcpy(normalized_P,d_p,sizeof(Elements_Chunk)*(fixed_len),hipMemcpyDeviceToHost);
		hipFree(d_chunk_P);
		hipFree(d_p);
		if(normalized_P != *P){
			free(*P);
			*P=normalized_P;
		}
	 }
/*
 * No need for multiexponents multiplication, it's the same as the FFT only with the multiexponents of another element.
 */
/*
 * inverse-Taylor expansion
 */

__device__ void itaylor_smaller_than_chunk(Chunk* chunk,idx_t in_chunk_idx, len_t t_dim, len_t p_len, idx_t idx){
		if(idx >= ((p_len) << (Chunk::log_cells_in_chunk)))
			return;
		//Performs the rest of the expansion.
		chunk_cell_t cell = chunk->v[in_chunk_idx];
		for(len_t i = 2; i<=t_dim && i <=Chunk::log_elements_in_chunk ; ++i){
			cell ^= (cell & (d_chunk_cell_mask[i]<<(1<<(i-1))))>>(1<<(i-2));
			cell ^= (cell & (d_chunk_cell_mask[i]<<((1<<i)-(1<<(i-2)))))>>(1<<(i-2));
		}
		chunk->v[in_chunk_idx]= cell;
}
__global__ void k_itaylorExpansion_iteration_large(Chunk* d_chunk_P , len_t p_len , len_t t_dim,len_t i){
	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	idx_t in_chunk_idx = idx & (Chunk::cells_in_chunk-1);
	/*
	 * flag = true if current sub-polynomial fits in a thread block.
	 */
	bool flag = (i-Chunk::log_elements_in_chunk <= log_max_block_size-Chunk::log_cells_in_chunk + 2);
	if(idx >= ((p_len) << (Chunk::log_cells_in_chunk-2)))
		return;
	do {
		len_t sub_len = (1<<(i-Chunk::log_elements_in_chunk));
		idx_t chunk_idx = idx >> (Chunk::log_cells_in_chunk);
		chunk_idx = (chunk_idx /(sub_len>>2))*(sub_len) + ((chunk_idx) & ((sub_len>>2)-1));
		d_chunk_P[(sub_len>>2) + chunk_idx ].v[in_chunk_idx] ^= d_chunk_P[(sub_len>>1) + chunk_idx].v[in_chunk_idx];
		d_chunk_P[(sub_len>>1) + chunk_idx ].v[in_chunk_idx] ^= d_chunk_P[(3*(sub_len>>2)) + chunk_idx].v[in_chunk_idx];
		if(flag)
			__syncthreads();
		++i;
	} while (i-Chunk::log_elements_in_chunk <= log_max_block_size-Chunk::log_cells_in_chunk + 2 && i<=t_dim);
		//If number of threads needed is less than a thread block - we can just continue!
}
__global__ void k_itaylorExpansion_iteration_twoChunks(Chunk* d_chunk_P, len_t p_len, len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	idx_t in_chunk_idx = idx & (Chunk::cells_in_chunk-1);
	if(idx >= ((p_len) << (Chunk::log_cells_in_chunk-1)))
		return;
	idx_t chunk_idx= (idx >> (Chunk::log_cells_in_chunk) ) << 1;
	d_chunk_P[(chunk_idx)].v[in_chunk_idx]^= (d_chunk_P[(chunk_idx)+1].v[in_chunk_idx]<<(Chunk::elements_in_chunk>>1));
	d_chunk_P[(chunk_idx)+1].v[in_chunk_idx]^= (d_chunk_P[(chunk_idx)+1].v[in_chunk_idx]>>(Chunk::elements_in_chunk>>1));

}
__global__ void k_itaylorExpansion_iteration_singleChunk(Chunk* d_chunk_P, len_t p_len, len_t t_dim){

	idx_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	idx_t in_chunk_idx = idx & (Chunk::cells_in_chunk-1);
	//Performs the rest of the expansion.
	idx_t chunk_idx = (idx >> (Chunk::log_cells_in_chunk));
	itaylor_smaller_than_chunk(d_chunk_P+chunk_idx,in_chunk_idx,t_dim,p_len,idx);
}
void itaylorExpansion_iteration(const FFT* fft, Chunk * d_chunk_P , len_t p_len , len_t t_dim,len_t i){
	unsigned int threads;
	dim3 blockSize(max_block_size,1,1);
	if( i >= Chunk::log_elements_in_chunk + 2){
		threads = p_len<<(Chunk::log_cells_in_chunk-2);
		dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
		k_itaylorExpansion_iteration_large<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim,i);
	} else if (i == Chunk::log_elements_in_chunk + 1){
		threads = p_len <<(Chunk::log_cells_in_chunk-1);
		dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
		k_itaylorExpansion_iteration_twoChunks<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
	} else {
		threads = p_len << Chunk::log_cells_in_chunk;
		dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
		k_itaylorExpansion_iteration_singleChunk<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
	}
//	k_taylorExpansion_iteration<<<gridSize,blockSize>>>(d_chunk_P,p_len,t_dim);
}
void GPU_FFT::itaylorExpansion_gpu(const FFT* fft, Chunk* d_chunk_P, len_t p_len , len_t t_dim){
	/*
	 * The sub-polynomial requires more than a single thread block.
	 */
	itaylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim,2);
	len_t i = Chunk::log_elements_in_chunk+1;
	if(t_dim>=i){
		itaylorExpansion_iteration(fft,d_chunk_P,p_len,i,i);
		++i;
	}
	if(t_dim>=i){
		itaylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim,i);
		i = log_max_block_size+Chunk::log_elements_in_chunk+3-Chunk::log_cells_in_chunk;
	}

	while(t_dim>=i){
		itaylorExpansion_iteration(fft,d_chunk_P,p_len,t_dim,i);
		++i;
	}
}
/*
 * inverse-partition
 */

	/*
	 *  Input:
	 *  1) d_chunk_p - The polynomial on device's memory.
	 *  2) p_len - number of chunks in d_chunk_p.
	 *  3) chunk_idx - the chunk current thread has to deal with.
	 *  4) in_chunk_idx - the number of the cell the current thread deals with.
	 *  5) t_dim - The dim of the original (input) subpolynomial.
	 *
	 *  This function performs what a single thread does when performing the partition function, on a single chunk.
	 */
__device__ void ipartition_in_chunk(Chunk* d_chunk_p, len_t p_len , idx_t in_chunk_idx, len_t t_dim){
	chunk_cell_t ans = d_chunk_p->v[in_chunk_idx];
	for(unsigned int i = MIN(t_dim,Chunk::log_elements_in_chunk) ; i >= 2 ; --i){
		ans = 	(ans & d_chunk_cell_mask[i]) |
				(ans & (d_chunk_cell_mask[i]<<((1<<(i))-(1<<(i-2))))) |
				(ans & (d_chunk_cell_mask[i]<<(1<<(i-1))))>>(1<<(i-2)) |
				(ans & (d_chunk_cell_mask[i]<<(1<<(i-2))))<<(1<<(i-2));
	}
	d_chunk_p->v[in_chunk_idx]=ans;
}

//__device__ void ipartition_two_chunks(Chunk* d_chunk_p, idx_t in_chunk_idx){
//	chunk_cell_t ans[2];
//	chunk_cell_t load[2];
//	const chunk_cell_t mask = d_chunk_cell_mask[Chunk::log_elements_in_chunk+1];
//	load[0]=d_chunk_p->v[in_chunk_idx];
//	load[1]=d_chunk_p[1].v[in_chunk_idx];
//	ans[0]=(load[0] & mask) | ((load[1] & mask)<<(Chunk::elements_in_chunk>>1));
//	ans[1]=(load[1] & (mask<< (Chunk::elements_in_chunk>>1)) )|
//			((load[0] >> (Chunk::elements_in_chunk>>1)) & mask);
//	d_chunk_p->v[in_chunk_idx] = ans[0];
//	d_chunk_p[1].v[in_chunk_idx] = ans[1];
//}

__device__ void ipartition_general(Chunk* d_chunk_p_src, Chunk* d_chunk_p_dst,  idx_t chunk_idx, idx_t in_chunk_idx, idx_t t_dim){
	if(chunk_idx & 1)
		d_chunk_p_dst[chunk_idx].v[in_chunk_idx] = d_chunk_p_src[(chunk_idx>>1) + (1<<(t_dim-1))].v[in_chunk_idx];
	else
		d_chunk_p_dst[chunk_idx].v[in_chunk_idx] = d_chunk_p_src[chunk_idx>>1].v[in_chunk_idx];
}

__global__ void k_ipartition_iteration_two_chunks(Chunk* d_chunk_p_src, len_t p_len){
	idx_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	idx_t chunk_idx = (idx >> Chunk::log_cells_in_chunk)<<1;
	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	if(chunk_idx >= p_len){
		return;
	}
	partition_two_chunks(d_chunk_p_src+chunk_idx, in_chunk_idx);
}
__global__ void k_ipartition_iteration_in_chunk(Chunk* d_chunk_p_src, len_t p_len, len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	idx_t chunk_idx = idx >> Chunk::log_cells_in_chunk;
	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	if(chunk_idx >= p_len){
		return;
	}
	ipartition_in_chunk(d_chunk_p_src+chunk_idx,p_len,in_chunk_idx,t_dim);
}
__global__ void k_ipartition_iteration_general(Chunk* d_chunk_p_src, Chunk* d_chunk_p_dst, len_t p_len, len_t t_dim){
	idx_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	idx_t chunk_idx = idx >> Chunk::log_cells_in_chunk;
	idx_t in_chunk_idx = idx & andMask(Chunk::log_cells_in_chunk);
	idx_t base = chunk_idx ^ (chunk_idx & andMask(t_dim-Chunk::log_elements_in_chunk));
	chunk_idx &= andMask(t_dim-Chunk::log_elements_in_chunk);
	if(base+chunk_idx >= p_len)
		return;
	ipartition_general(d_chunk_p_src+base,d_chunk_p_dst+base, chunk_idx, in_chunk_idx,t_dim-Chunk::log_elements_in_chunk );
}
/*
 * If t_dim > Chunk::log_elements_in_chunk+1 the result is written in dst, otherwise it will be written in src.
 */
bool GPU_FFT::ipartition(Chunk* d_chunk_p_src, Chunk* d_chunk_p_dst, len_t p_len, len_t t_dim){
	len_t threads = p_len << Chunk::log_cells_in_chunk;
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
	bool flag = false;
	if(t_dim > Chunk::log_elements_in_chunk+1){
		k_ipartition_iteration_general<<<gridSize,blockSize>>>(d_chunk_p_src,d_chunk_p_dst,p_len,t_dim);
		Chunk* swap;
		swap = d_chunk_p_src;
		d_chunk_p_src = d_chunk_p_dst;
		d_chunk_p_dst = swap;
		flag= true;
	}
	if(t_dim >= Chunk::log_elements_in_chunk +1){
      	dim3 gridSizeTwoChunks(sizeCiel(threads>>1,max_block_size),1,1);
		k_ipartition_iteration_two_chunks<<<gridSizeTwoChunks,blockSize>>>(d_chunk_p_src,p_len);
	}
	k_ipartition_iteration_in_chunk<<<gridSize,blockSize>>>(d_chunk_p_src,p_len,t_dim);
	return flag;
}
/*
 * inverse linear evaluation
 */
__global__ void k_icopy_and_shift_vec(Chunk* d_chunk_src, Chunk* d_chunk_dst, len_t p_len)
{
	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
	__shared__ Chunk c_shared[shared_len<<1];
	chunk_cell_t t;
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);

	if(chunk_idx  >= p_len){
		return;
	}

	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);

	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
		my_shared_chunk->v[in_chunk_idx+i]=d_chunk_src[chunk_idx].v[in_chunk_idx+i];
		my_shared_chunk[1].v[in_chunk_idx+i]=d_ilinear_mul->v[in_chunk_idx+i];
	}

	chunk_cell_t tmp;
#pragma unroll
	for(unsigned int i = 0 ; i < Element::ord ; i+=warp_size){
		tmp = my_shared_chunk->v[in_chunk_idx+i];
		t = (tmp& 0x55555555) ^ ((tmp & 0xaaaaaaaa)>>1);
		my_shared_chunk->v[in_chunk_idx+i] = t^(t<<1);
		d_chunk_dst[chunk_idx].v[in_chunk_idx+i] = (tmp & 0x55555555) ;
	}
	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
	Chunk::chunk_reduce(d_chunk_src+chunk_idx,my_shared_chunk,in_chunk_idx);
	for(unsigned int i = 0 ; i < Element::ord ; i+=warp_size){
		d_chunk_src[chunk_idx].v[in_chunk_idx+i] ^= d_chunk_dst[chunk_idx].v[in_chunk_idx+i];
	}
	return;
}
void GPU_FFT::ilinearEvaluation(Chunk* d_chunk_p,Chunk* d_chunk_p_cpy, len_t p_len)
{
	len_t threads = p_len << Chunk::log_threads_in_chunk;
	dim3 blockSize(max_block_size,1,1);
	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
	k_icopy_and_shift_vec<<<gridSize,blockSize>>>(d_chunk_p,d_chunk_p_cpy,p_len);
}
/*
 * UVFromW
 */

__global__ void k_isubspaceMult_chunk_and_add(Chunk* d_a, Chunk* d_b , len_t log_elements_in_b ,len_t a_len ){
	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
	if(chunk_idx  >= a_len){
		return;
	}
	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
	Chunk::clmul_by_chunk_bShuffle_ixor_mask(d_a[chunk_idx],*d_b,in_chunk_idx,d_a[chunk_idx],1<<log_elements_in_b,log_elements_in_b);

}
/*
 * Dim = Size of U.
 */
void GPU_FFT::UVFromW(Chunk* d_chunk_p, len_t p_len, Chunk* subspace, len_t dim){
	if(dim>=Chunk::log_elements_in_chunk){
		len_t threadsMul = p_len << (Chunk::log_threads_in_chunk-1);
		len_t threadsAdd = p_len << (Chunk::log_cells_in_chunk-1);
		dim3 block(multThreadsInBlock,1,1);
		dim3 gridMul(sizeCiel(threadsMul,multThreadsInBlock),1,1);
		dim3 gridAdd(sizeCiel(threadsAdd,multThreadsInBlock),1,1);
		k_subspaceAdd_general<<<gridAdd,block>>>(d_chunk_p,p_len,1<<(dim-Chunk::log_elements_in_chunk));
		k_subspaceMult_general<<<gridMul,block>>>(d_chunk_p, subspace, (1<<(dim-Chunk::log_elements_in_chunk)),p_len);
	} else {
		len_t threadsMul = p_len << Chunk::log_threads_in_chunk;
		dim3 block(multThreadsInBlock,1,1);
		dim3 gridMul(sizeCiel(threadsMul,multThreadsInBlock),1,1);
		k_isubspaceMult_chunk_and_add<<<gridMul,block>>>(d_chunk_p,subspace, dim , p_len);
	}
}

}

#endif
